#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <builtin_types.h>

#include <GL/glew.h>
#include <GLFW/glfw3.h>

#include "scatterlife.h" 


// CUDA runtime
//#include <hip/hip_cooperative_groups.h>
//using namespace cooperative_groups;


GLuint rasterTexture;

GLFWwindow* window;


void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
    glViewport(0, 0, width, height);
}









//=========================================
// BEGIN CUDA KERNELS
//=========================================






__device__ Universe univ = {};
__device__ Universe univ2 = {};


__device__ UniImg raster = {};

__device__ volatile unsigned int gTime = 1; 

//__device__ volatile unsigned int maxParticleCount = 0;


Universe host_univ = {};
UniImg host_raster = {};





// hexagonal particle storage
// 0 N
// 1 NE
// 2 SE
// 3 S
// 4 SW
// 5 NW

__global__ void runAutomata(bool direction){

  Universe* origin;
  Universe* target;
  if (direction){
    origin = &univ;
    target = &univ2;
  }else{
    origin = &univ2;
    target = &univ;
  }

  unsigned int x = blockIdx.x;
  unsigned int y = blockIdx.y;

  unsigned int seed = gTime; //+ x + y + x*y + y*x*x+ y*y*x;

  unsigned int xm1 = x >= 1 ? x-1 : (UNIVERSE_WIDTH - 1);
  unsigned int xp1 = x < (UNIVERSE_WIDTH - 1) ? x+1 : 0;

  unsigned int ym1 = y >= 1 ? y-1 : (UNIVERSE_HEIGHT - 1);
  unsigned int yp1 = y < (UNIVERSE_HEIGHT - 1) ? y+1 : 0;


  unsigned int incoming[6] = {
    (*origin)   [x] [ym1]  .unbound[0],   
    (*origin) [xp1] [ym1]  .unbound[1],
    (*origin) [xp1]   [y]  .unbound[2],   
    (*origin)   [x] [yp1]  .unbound[3],   
    (*origin) [xm1] [yp1]  .unbound[4], 
    (*origin) [xm1]   [y]  .unbound[5]   
  };

  //this_grid().sync();

  unsigned int triforce_a = min(min(incoming[0], incoming[2]), incoming[4]);
  unsigned int triforce_b = min(min(incoming[1], incoming[3]), incoming[5]);

  unsigned int pair_a = min(incoming[0], incoming[3]);
  unsigned int pair_b = min(incoming[1], incoming[4]);
  unsigned int pair_c = min(incoming[2], incoming[5]);
  
  unsigned int pairTriforceOverlap = min( 
    max(max(pair_a, pair_b), pair_c),
    max(triforce_a, triforce_b)
  );

  unsigned int pairsKept = seed % (pairTriforceOverlap + 1);
  unsigned int triforcesKept = pairTriforceOverlap - pairsKept;

  pair_a -= min(pair_a, triforcesKept);
  pair_b -= min(pair_b, triforcesKept);
  pair_c -= min(pair_c, triforcesKept);

  unsigned int final_pairs = pair_a + pair_b + pair_c;

  triforce_a -= min(triforce_a, pairsKept);
  triforce_b -= min(triforce_b, pairsKept);


  unsigned int final_triforces = triforce_a + triforce_b;
  

  

  unsigned int triforce_cut = seed % (final_triforces + 1);

  unsigned int final_triforce_a = triforce_cut;
  unsigned int final_triforce_b = final_triforces - triforce_cut;

  // ordered pairing function
  unsigned int z = seed % (
     (final_pairs + 1)*(final_pairs + 2)/2
  );

  unsigned int w = (sqrtf(8*z + 1) - 1) / 2;
  unsigned int pair_cut_1 = z - w*(w+1)/2;
  unsigned int pair_cut_2 = w;

  unsigned int final_pair_a = pair_cut_1;
  unsigned int final_pair_b = pair_cut_2 - pair_cut_1;
  unsigned int final_pair_c = final_pairs - pair_cut_2;



  unsigned int scattering[6] = {
    final_triforce_a + final_pair_a,
    final_triforce_b + final_pair_b,
    final_triforce_a + final_pair_c,
    final_triforce_b + final_pair_a,
    final_triforce_a + final_pair_b,
    final_triforce_b + final_pair_c
  };




  

  
  (*target)[x][y] = {
    scattering[0],
    scattering[1],
    scattering[2],
    scattering[3],
    scattering[4],
    scattering[5],
    (*origin)[x][y].bound[0] + incoming[0] - triforce_a - pair_a,
    (*origin)[x][y].bound[1] + incoming[1] - triforce_b - pair_b,
    (*origin)[x][y].bound[2] + incoming[2] - triforce_a - pair_c,
    (*origin)[x][y].bound[3] + incoming[3] - triforce_b - pair_a,
    (*origin)[x][y].bound[4] + incoming[4] - triforce_a - pair_b,
    (*origin)[x][y].bound[5] + incoming[5] - triforce_b - pair_c,
  };

  if (blockIdx.x == 0){
    ++gTime;
  }
}

__global__ void rasterizeAutomata(){
  unsigned int x = blockIdx.x;
  unsigned int y = blockIdx.y;

  // if (blockIdx.x == 0){
  //   maxParticleCount = 0;
  // }

  // unsigned int pc = (
  //     univ[x][y].bound[0] + univ[x][y].unbound[0]
  //   + univ[x][y].bound[1] + univ[x][y].unbound[1]
  //   + univ[x][y].bound[2] + univ[x][y].unbound[2]
  //   + univ[x][y].bound[3] + univ[x][y].unbound[3]
  //   + univ[x][y].bound[4] + univ[x][y].unbound[4]
  //   + univ[x][y].bound[5] + univ[x][y].unbound[5]
  // );

  //atomicMax((unsigned int*) &maxParticleCount, pc);

  //this_grid().sync();

  //AAGGBBRR
  raster[x][y] = 
      univ[x][y].bound[0] || univ[x][y].unbound[0]
    || univ[x][y].bound[1] || univ[x][y].unbound[1]
    || univ[x][y].bound[2] || univ[x][y].unbound[2]
    || univ[x][y].bound[3] || univ[x][y].unbound[3]
    || univ[x][y].bound[4] || univ[x][y].unbound[4]
    || univ[x][y].bound[5] || univ[x][y].unbound[5] ? RGB{1.0f, 1.0f, 1.0f} : RGB{0.0f, 0.0f, 0.0f};

  //(unsigned int)(16777215.0 * powf(pc / maxParticleCount, 0.2)) | 0xFF000000;
}



//=========================================
// END CUDA KERNELS
//=========================================








#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPU Error: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}




void dump_univ(){
  for (int i = 0; i < UNIVERSE_WIDTH; ++i){
    for (int z = 0; z < UNIVERSE_HEIGHT; ++z){
      printf("%3d", host_raster[i][z]);
    }
    printf("\n");
  }
}


void initOpenGL(){

  glfwInit();

  const GLFWvidmode* mode = glfwGetVideoMode(glfwGetPrimaryMonitor());

  // glfwWindowHint(GLFW_RED_BITS, mode->redBits);
  // glfwWindowHint(GLFW_GREEN_BITS, mode->greenBits);
  // glfwWindowHint(GLFW_BLUE_BITS, mode->blueBits);
  // glfwWindowHint(GLFW_REFRESH_RATE, mode->refreshRate);

  int width = mode->width*.75;//UNIVERSE_WIDTH;
  int height = mode->height*.75;//UNIVERSE_HEIGHT;
  window = glfwCreateWindow(width, height, "ScatterLife", NULL, NULL);
  glfwSetWindowPos(window, (mode->width - width) / 2, (mode->height - height) / 2);

  glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);

  glfwMakeContextCurrent(window);

  glewInit();

  // setup raster to texture modes
  glPixelStorei(GL_UNPACK_ALIGNMENT, 1);
  glGenTextures(1, &rasterTexture);   // generate a texture handler really reccomanded (mandatory in openGL 3.0)
  glBindTexture(GL_TEXTURE_2D, rasterTexture); // tell openGL that we are using the texture 

  glEnable(GL_TEXTURE_2D);

  // glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
  // glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);

  // glTexParameteri(GL_TEXTURE_2D, GL_GENERATE_MIPMAP, GL_TRUE);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, 
                 GL_LINEAR);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, 
                 GL_LINEAR);
  GLfloat fLargest;
  glGetFloatv(GL_MAX_TEXTURE_MAX_ANISOTROPY, &fLargest);
  glTexParameterf(GL_TEXTURE_2D, GL_TEXTURE_MAX_ANISOTROPY, fLargest);

  glMatrixMode(GL_PROJECTION);

  GLdouble matrix[16] = {
    3.0/2.0, 0, 0, 0,
    sqrt(3.0)/2.0, sqrt(3.0), 0, 0,
    0, 0, 1, 0,
    0, 0, 0, 1
  };
  glLoadMatrixd(matrix);
  //glOrtho(0.0f, UNIVERSE_WIDTH, UNIVERSE_HEIGHT, 0.0f, 0.0f, 1.0f);
  //glEnable(GL_BLEND);
  //glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
}



int main(int argc, char **argv)
{
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  printf("  Device name: %s\n", prop.name);

  hipSetDevice(0);

  // //initialize automata
  // for (int x = 0; x < UNIVERSE_WIDTH; ++x){
  //   for (int y = 0; y < UNIVERSE_HEIGHT; ++y){
  //     host_univ[x][y] = 
  //   }
  // }
  
  //initialize INITIAL_PARTICLE_COUNT heading to center cell from every neighbor
  //host_univ[UNIVERSE_WIDTH/2][UNIVERSE_HEIGHT/2-1].unbound[0] = INITIAL_PARTICLE_COUNT;
  //host_univ[UNIVERSE_WIDTH/2+1][UNIVERSE_HEIGHT/2-1].unbound[1] = INITIAL_PARTICLE_COUNT;
  //host_univ[UNIVERSE_WIDTH/2+1][UNIVERSE_HEIGHT/2].unbound[2] = INITIAL_PARTICLE_COUNT;
  //host_univ[UNIVERSE_WIDTH/2][UNIVERSE_HEIGHT/2+1].unbound[3] = INITIAL_PARTICLE_COUNT;
  //host_univ[UNIVERSE_WIDTH/2-1][UNIVERSE_HEIGHT/2+1].unbound[4] = INITIAL_PARTICLE_COUNT;
  //host_univ[UNIVERSE_WIDTH/2-1][UNIVERSE_HEIGHT/2].unbound[5] = INITIAL_PARTICLE_COUNT;

  //also set a 2nd wave of INITIAL_PARTICLE_COUNT heading to center cell from every neighbor
  host_univ[UNIVERSE_WIDTH/2][UNIVERSE_HEIGHT/2-1].bound[0] = INITIAL_PARTICLE_COUNT;
  //host_univ[UNIVERSE_WIDTH/2+1][UNIVERSE_HEIGHT/2-1].bound[1] = INITIAL_PARTICLE_COUNT;
  //host_univ[UNIVERSE_WIDTH/2+1][UNIVERSE_HEIGHT/2].bound[2] = INITIAL_PARTICLE_COUNT;
  host_univ[UNIVERSE_WIDTH/2][UNIVERSE_HEIGHT/2+1].bound[3] = INITIAL_PARTICLE_COUNT;
  //host_univ[UNIVERSE_WIDTH/2-1][UNIVERSE_HEIGHT/2+1].bound[4] = INITIAL_PARTICLE_COUNT;
  //host_univ[UNIVERSE_WIDTH/2-1][UNIVERSE_HEIGHT/2].bound[5] = INITIAL_PARTICLE_COUNT;

  hipMemcpyToSymbol(HIP_SYMBOL(univ), host_univ, sizeof(Universe), 0, hipMemcpyHostToDevice);

  
  



  initOpenGL();

  float scale = 0.4f;

  char title[128];
  hipEvent_t start, stop;
  float milliseconds = 0;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  /* Loop until the user closes the window */
  while (!glfwWindowShouldClose(window))
  {
      hipEventRecord(start);
      runAutomata<<<dim3(UNIVERSE_WIDTH, UNIVERSE_HEIGHT, 1), dim3(1,1,1)>>>(true);
      runAutomata<<<dim3(UNIVERSE_WIDTH, UNIVERSE_HEIGHT, 1), dim3(1,1,1)>>>(false);
      hipEventRecord(stop);

      // rasterize
      rasterizeAutomata<<<dim3(UNIVERSE_WIDTH, UNIVERSE_HEIGHT, 1), dim3(1,1,1)>>>();

      // copy raster back to host
      hipMemcpyFromSymbol(host_raster, HIP_SYMBOL(raster), sizeof(UniImg), 0, hipMemcpyDeviceToHost);

      //hipStreamSynchronize(hipStreamPerThread);

      //glClear(GL_COLOR_BUFFER_BIT);
      
      glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, RASTER_WIDTH, RASTER_HEIGHT, 0, GL_RGB, GL_FLOAT, host_raster);
      //glGenerateTextureMipmap(rasterTexture);

      glBegin(GL_TRIANGLE_STRIP);

      glTexCoord2f(1.0f, 1.0f); glVertex2f(-scale, -scale);
      glTexCoord2f(1.0f, 0.0f); glVertex2f(-scale, scale);
      glTexCoord2f(0.0f, 1.0f); glVertex2f(scale, -scale);
      glTexCoord2f(0.0f, 0.0f); glVertex2f(scale, scale);

      glEnd();

      glfwSwapBuffers(window);


      hipEventElapsedTime((float*)&milliseconds, start, stop);
      sprintf(title, "%.2f executions per sec",  2000.0f / (float) milliseconds);

      glfwSetWindowTitle(window, title);

      glfwPollEvents();

      //Sleep(500);
  }
}